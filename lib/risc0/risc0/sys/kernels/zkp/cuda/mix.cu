#include "hip/hip_runtime.h"
// Copyright 2022 Risc0, Inc.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "fp.h"
#include "fp4.h"

extern "C" __global__
void mix_poly_coeffs(Fp4* out,
                     const Fp* in,
                     const uint32_t* combos,
                     const Fp4& mixStart,
                     const Fp4& mix,
                     const uint32_t inputSize,
                     const uint32_t count) {
  uint idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < count) {
    Fp4 cur = mixStart;
    for (size_t i = 0; i < inputSize; i++) {
      size_t id = combos[i];
      out[count * id + idx] += cur * in[count * i + idx];
      cur *= mix;
    }
  }
}
