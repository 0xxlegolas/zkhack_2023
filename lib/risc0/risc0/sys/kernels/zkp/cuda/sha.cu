#include "hip/hip_runtime.h"
// Copyright 2022 Risc0, Inc.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "sha256.h"

extern "C" __global__
void sha_rows(ShaDigest* out,
              const Fp* matrix,
              uint32_t count,
              uint32_t colSize) {
  uint32_t idx = blockDim.x * blockIdx.x + threadIdx.x;
  if (idx < count) {
    out[idx] = shaHash(matrix + idx, colSize, count, false);
  }
}

extern "C" __global__
void sha_fold(ShaDigest* out,
              const ShaDigest* in,
              uint32_t count) {
  uint32_t idx = blockDim.x * blockIdx.x + threadIdx.x;
  if (idx < count) {
    out[idx] = shaHashPair(in[2 * idx], in[2 * idx + 1]);
  }
}
