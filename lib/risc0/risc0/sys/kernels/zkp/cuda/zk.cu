#include "hip/hip_runtime.h"
// Copyright 2022 Risc0, Inc.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "fp.h"

extern "C" __global__
void zk_shift(Fp* io,
              const uint32_t bits,
              const uint32_t count) {
  uint idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < count) {
    uint32_t pos = idx & ((1 << bits) - 1);
    uint32_t posRev = __brev(pos) >> (32 - bits);
    Fp pow3 = pow(Fp(3), posRev);
    io[idx] = io[idx] * pow3;
  }
}
