#include "hip/hip_runtime.h"
// Copyright 2022 Risc0, Inc.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "fp.h"
#include "fp4.h"

extern "C" __global__
void multi_bit_reverse(Fp* io,
                       const uint32_t nBits,
                       const uint32_t count) {
  uint totIdx = blockIdx.x * blockDim.x + threadIdx.x;
  if (totIdx < count) {
    uint32_t rowSize = 1 << nBits;
    uint32_t idx = totIdx & (rowSize - 1);
    uint32_t s = totIdx >> nBits;
    uint32_t ridx = __brev(idx) >> (32 - nBits);
    if (idx < ridx) {
      size_t idx1 = s * rowSize + idx;
      size_t idx2 = s * rowSize + ridx;
      Fp tmp = io[idx1];
      io[idx1] = io[idx2];
      io[idx2] = tmp;
    }
  }
}

extern "C" __global__
void multi_poly_eval(Fp4* out,
                     const Fp* coeffs,
                     const uint32_t* which,
                     const Fp4* xs,
                     const uint32_t deg) {
  const Fp* cur_poly = coeffs + which[blockIdx.x] * deg;
  Fp4 x = xs[blockIdx.x];
  Fp4 stepx = pow(x, blockDim.x);
  Fp4 powx = pow(x, threadIdx.x);
  Fp4 tot;
  for (size_t i = threadIdx.x; i < deg; i += blockDim.x) {
    tot += powx * cur_poly[i];
    powx *= stepx;
  }
  extern __shared__ uint32_t totsBuf[];
  Fp4* tots = reinterpret_cast<Fp4*>(totsBuf);
  tots[threadIdx.x] = tot;
  __syncthreads();
  unsigned cur = blockDim.x;
  while (cur) {
    cur /= 2;
    if (threadIdx.x < cur) {
      tots[threadIdx.x] = Fp4(tots[threadIdx.x]) + Fp4(tots[threadIdx.x + cur]);
    }
    __syncthreads();
  }
  if (threadIdx.x == 0) {
    out[blockIdx.x] = tots[0];
  }
}

extern "C" __global__
void batch_expand(Fp* out,
                  const Fp* in,
                  const uint32_t polyCount,
                  const uint32_t outSize,
                  const uint32_t inSize,
                  const uint32_t expandBits) {
  uint idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < outSize) {
    for (uint32_t i = 0; i < polyCount; i++) {
      out[i * outSize + idx] = in[i * inSize + (idx >> expandBits)];
    }
  }
}

extern "C" __global__
void gather_sample(Fp* dst,
                   const Fp* src,
                   const uint32_t idx,
                   const uint32_t size,
                   const uint32_t stride) {
  uint gid = blockIdx.x * blockDim.x + threadIdx.x;
  if (gid < size) {
      dst[gid] = src[gid * stride + idx];
  }
}

extern "C" __global__
void multi_ntt_fwd_step(Fp* io,
                        const Fp* rou,
                        const uint32_t nBits,
                        const uint32_t sBits,
                        const uint32_t cSize) {
  uint32_t gSize = 1 << (nBits - sBits);
  uint32_t sSize = 1 << (sBits - 1);
  uint32_t nSize = 1 << nBits;
  uint32_t sOff = threadIdx.x + blockIdx.x * blockDim.x;
  uint32_t sStep = blockDim.x * gridDim.x;
  uint32_t gOff = threadIdx.y + blockIdx.y * blockDim.y;
  uint32_t gStep = blockDim.y * gridDim.y;
  uint32_t cOff = threadIdx.z + blockIdx.z * blockDim.z;
  uint32_t cStep = blockDim.z * gridDim.z;

  // Compute the intial multiplier for the sOff: pow(rou[s], sOff)
  Fp curMul(1);
  uint32_t curRou = sBits;
  uint32_t powX = sOff;
  while (curRou > 0) {
    if (powX & 1) {
      curMul = curMul * rou[curRou];
    }
    powX >>= 1;
    curRou--;
  }
  // Compute the multiplier for each loop around s
  int rouStep = __ffs(sSize / sStep);
  Fp stepMul = rou[rouStep];
  for (uint32_t s = sOff; s < sSize; s += sStep) {
    for (uint32_t g = gOff; g < gSize; g += gStep) {
      for (uint32_t c = cOff; c < cSize; c += cStep) {
        Fp a = io[c * nSize + g * 2 * sSize + s];
        Fp b = io[c * nSize + g * 2 * sSize + s + sSize];
        b *= curMul;
        io[c * nSize + g * 2 * sSize + s] = a + b;
        io[c * nSize + g * 2 * sSize + s + sSize] = a - b;
      }
    }
    curMul *= stepMul;
  }
}

extern "C" __global__
void multi_ntt_rev_step(Fp* io,
                        const Fp* rou,
                        const uint32_t nBits,
                        const uint32_t sBits,
                        const uint32_t cSize) {
  uint32_t gSize = 1 << (nBits - sBits);
  uint32_t sSize = 1 << (sBits - 1);
  uint32_t nSize = 1 << nBits;
  uint32_t sOff = threadIdx.x + blockIdx.x * blockDim.x;
  uint32_t sStep = blockDim.x * gridDim.x;
  uint32_t gOff = threadIdx.y + blockIdx.y * blockDim.y;
  uint32_t gStep = blockDim.y * gridDim.y;
  uint32_t cOff = threadIdx.z + blockIdx.z * blockDim.z;
  uint32_t cStep = blockDim.z * gridDim.z;

  // Compute the intial multiplier for the sOff: pow(rou[s], sOff)
  Fp curMul(1);
  uint32_t curRou = sBits;
  uint32_t powX = sOff;
  while (curRou > 0) {
    if (powX & 1) {
      curMul = curMul * rou[curRou];
    }
    powX >>= 1;
    curRou--;
  }
  // Compute the multiplier for each loop around s
  int rouStep = __ffs(sSize / sStep);
  Fp stepMul = rou[rouStep];
  for (uint32_t s = sOff; s < sSize; s += sStep) {
    for (uint32_t g = gOff; g < gSize; g += gStep) {
      for (uint32_t c = cOff; c < cSize; c += cStep) {
        Fp a = io[c * nSize + g * 2 * sSize + s];
        Fp b = io[c * nSize + g * 2 * sSize + s + sSize];
        io[c * nSize + g * 2 * sSize + s] = a + b;
        io[c * nSize + g * 2 * sSize + s + sSize] = (a - b) * curMul;
      }
    }
    curMul *= stepMul;
  }
}
