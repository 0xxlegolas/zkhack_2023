#include "hip/hip_runtime.h"
// Copyright 2023 Risc0, Inc.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "fp.h"

#define CELLS 24
#define ROUNDS_FULL 8
#define ROUNDS_HALF_FULL (ROUNDS_FULL / 2)
#define ROUNDS_PARTIAL 21
#define ROW_SIZE (CELLS + ROUNDS_PARTIAL)
#define CELLS_RATE 16
#define CELLS_OUT 8

__device__ void add_round_constants(const Fp* ROUND_CONSTANTS, Fp* cells, uint round) {
  for (uint i = 0; i < CELLS; i++) {
    cells[i] += ROUND_CONSTANTS[round * CELLS + i];
  }
}

__device__ Fp sbox(Fp x) {
  Fp x2 = x * x;
  Fp x4 = x2 * x2;
  Fp x6 = x4 * x2;
  return x6 * x;
}

__device__ void do_full_sboxes(Fp* cells) {
  for (uint i = 0; i < CELLS; i++) {
    cells[i] = sbox(cells[i]);
  }
}

__device__ void multiply_by_mds(const Fp* MDS, Fp* cells) {
  Fp new_cells[CELLS];
  for (uint i = 0; i < CELLS; i++) {
    Fp tot = 0;
    for (uint j = 0; j < CELLS; j++) {
      tot += MDS[i * CELLS + j] * cells[j];
    }
    new_cells[i] = tot;
  }
  for (uint i = 0; i < CELLS; i++) {
    cells[i] = new_cells[i];
  }
}

__device__ void full_round(const Fp* ROUND_CONSTANTS, const Fp* MDS, Fp* cells, uint round) {
    add_round_constants(ROUND_CONSTANTS, cells, round);
    do_full_sboxes(cells);
    multiply_by_mds(MDS, cells);
}

__device__ void poseidon_mix(const Fp* ROUND_CONSTANTS,
                const Fp* MDS,
                const Fp* PARTIAL_COMP_MATRIX,
                const Fp* PARTIAL_COMP_OFFSET,
                Fp* cells) {
  uint round = 0;
  for (uint i = 0; i < ROUNDS_HALF_FULL; i++) {
    full_round(ROUND_CONSTANTS, MDS, cells, round);
    round++;
  }
  Fp sboxes[ROUNDS_PARTIAL];
  for (uint i = 0; i < ROUNDS_PARTIAL; i++) {
    // For each sbox, compute it's input
    Fp sbox_in = PARTIAL_COMP_OFFSET[CELLS + i];
    for (uint j = 0; j < CELLS; j++) {
      sbox_in += PARTIAL_COMP_MATRIX[(CELLS + i) * ROW_SIZE + j] * cells[j];
    }
    for (uint j = 0; j < i; j++) {
      sbox_in += PARTIAL_COMP_MATRIX[(CELLS + i) * ROW_SIZE + CELLS + j] * sboxes[j];
    }
    // Run it through the sbox + record it
    sboxes[i] = sbox(sbox_in);
  }
  // Forward output data back to cells
  Fp new_cells[CELLS];
  for (uint i = 0; i < CELLS; i++) {
    Fp out = PARTIAL_COMP_OFFSET[i];
    for (uint j = 0; j < CELLS; j++) {
      out += PARTIAL_COMP_MATRIX[i * ROW_SIZE + j] * cells[j];
    }
    for (uint j = 0; j < ROUNDS_PARTIAL; j++) {
      out += PARTIAL_COMP_MATRIX[i * ROW_SIZE + CELLS + j] * sboxes[j];
    }
    new_cells[i] = out;
  }
  round += ROUNDS_PARTIAL;
  for (uint i = 0; i < CELLS; i++) {
    cells[i] = new_cells[i];
  }
  for (uint i = 0; i < ROUNDS_HALF_FULL; i++) {
    full_round(ROUND_CONSTANTS, MDS, cells, round);
    round++;
  }
}

extern "C" __global__ void poseidon_fold(const Fp* ROUND_CONSTANTS,
                     const Fp* MDS,
                     const Fp* PARTIAL_COMP_MATRIX,
                     const Fp* PARTIAL_COMP_OFFSET,
                     Fp* output,
                     const Fp* input,
                     uint32_t output_size) {
  uint32_t gid = blockDim.x * blockIdx.x + threadIdx.x;
  if (gid >= output_size) { return; }
  Fp cells[CELLS];
  for (size_t i = 0; i < CELLS_OUT; i++) {
    cells[i] = input[2 * gid * CELLS_OUT + i];
    cells[CELLS_OUT + i] = input[(2 * gid + 1) * CELLS_OUT + i];
  }
  poseidon_mix(ROUND_CONSTANTS, MDS, PARTIAL_COMP_MATRIX, PARTIAL_COMP_OFFSET, cells);
  for (uint i = 0; i < CELLS_OUT; i++) {
    output[gid * CELLS_OUT + i] = cells[i];
  }
}

extern "C" __global__ void poseidon_rows(const Fp* ROUND_CONSTANTS,
                     const Fp* MDS,
                     const Fp* PARTIAL_COMP_MATRIX,
                     const Fp* PARTIAL_COMP_OFFSET,
                     Fp* out,
                     const Fp* matrix,
                     uint32_t count,
                     uint32_t col_size) { 
  uint32_t gid = blockDim.x * blockIdx.x + threadIdx.x;
  if (gid >= count) { return; }
  Fp cells[CELLS];
  uint used = 0;
  for (uint i = 0; i < col_size; i++) {
    cells[used++] += matrix[i * count + gid];
    if (used == CELLS_RATE) {
      poseidon_mix(ROUND_CONSTANTS, MDS, PARTIAL_COMP_MATRIX, PARTIAL_COMP_OFFSET, cells);
      used = 0;
    }
  }
  if (used != 0 || count == 0) {
    poseidon_mix(ROUND_CONSTANTS, MDS, PARTIAL_COMP_MATRIX, PARTIAL_COMP_OFFSET, cells);
  }
  for (uint i = 0; i < CELLS_OUT; i++) {
    out[CELLS_OUT * gid + i] = cells[i];
  }
}

