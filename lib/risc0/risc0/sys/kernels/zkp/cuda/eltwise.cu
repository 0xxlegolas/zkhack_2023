#include "hip/hip_runtime.h"
// Copyright 2022 Risc0, Inc.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "fp.h"
#include "fp4.h"

extern "C" __global__
void eltwise_add_fp(Fp* out,
                    const Fp* x,
                    const Fp* y,
                    const uint32_t count) {
  uint idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < count) {
    out[idx] = x[idx] + y[idx];
  }
}

extern "C" __global__
void eltwise_mul_factor_fp(Fp* io,
                           const Fp& factor,
                           const uint32_t count) {
  uint idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < count) {
    io[idx] = io[idx] * factor;
  }
}

extern "C" __global__
void eltwise_copy_fp(Fp* out,
                     const Fp* in,
                     const uint32_t count) {
  uint idx = blockIdx.x * blockDim.x + threadIdx.x;
  // If the following check is not included, there is a SIGABRT that causes tests to fail
  // cuda-memcheck also throws lots of out of bounds read errors if this check is omitted
  if (idx < count) {
    out[idx] = in[idx];
  }
}

extern "C" __global__
void eltwise_sum_fp4(Fp* out,
                     const Fp4* in,
                     const uint32_t to_add,
                     const uint32_t count) {
  uint idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < count) {
    Fp4 tot;
    for (size_t i = 0; i < to_add; i++) {
      tot += in[count * i + idx];
    }
    out[idx + 0 * count] = tot.elems[0];
    out[idx + 1 * count] = tot.elems[1];
    out[idx + 2 * count] = tot.elems[2];
    out[idx + 3 * count] = tot.elems[3];
  }
}
