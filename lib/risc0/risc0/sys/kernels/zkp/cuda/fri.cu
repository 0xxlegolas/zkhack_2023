#include "hip/hip_runtime.h"
// Copyright 2022 Risc0, Inc.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "fp.h"
#include "fp4.h"

constexpr size_t kFriFold = 16;

/// Compute `ceil(log_2(in))`, i.e. find the smallest value `out` such that `2^out >= in`.
__device__ inline constexpr size_t log2Ceil(size_t in) {
  size_t r = 0;
  while ((1 << r) < in) {
    r++;
  }
  return r;
}

extern "C" __global__
void fri_fold(Fp* out,
              const Fp* in,
              const Fp4& mix,
              const uint32_t count) {
  uint idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < count) {
    Fp4 tot;
    Fp4 curMix(1);
    for (uint32_t i = 0; i < kFriFold; i++) {
      size_t rev_i = __brev(i) >> (32 - log2Ceil(kFriFold));
      size_t rev_idx = rev_i * count + idx;
      Fp4 factor(in[0 * count * kFriFold + rev_idx],
                 in[1 * count * kFriFold + rev_idx],
                 in[2 * count * kFriFold + rev_idx],
                 in[3 * count * kFriFold + rev_idx]);
      tot += curMix * factor;
      curMix *= mix;
    }
    for (size_t i = 0; i < 4; i++) {
      out[count * i + idx] = tot.elems[i];
    }
  }
}
